#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
// discretization in m
#define dx 0.001
//x, y and z dimensiona
#define X_SIZE 100
#define Y_SIZE 100
#define Z_SIZE 100
#define DIM 3
// volume em metros cúbicos
#define volume dx*(X_SIZE-1)*dx*(Y_SIZE-1)*dx*(Z_SIZE-1)
//densidade de massa (kg/m^3):
#define ro 100
// a massa é encontrada multiplicando o volume pela densidade
#define mass volume*ro
// força máxima por área, em pascal (N/m^2)
#define pressure 13500
#define HEALTHY 0
#define ISCHEMIC 1
#define DEAD	3
//força máxima que um elemento faz ao ser ativada pelo AP -> multiplica a pressão pela área formada pelo quadrado (dx X dx) e divide por quatro, representando os 4 elmentos q formam um quadrado
#define tForce pressure*(dx*dx)/4
#define get1DIndex(i, j, k) i*X_SIZE*Y_SIZE + j*Y_SIZE + k
// spring coefficient N/m
#define ks 15.0
// Overdamped (ζ > 1): The system returns (exponentially decays) to equilibrium without oscillating. Larger values of the damping ratio ζ return to equilibrium more slowly.
// Critically damped (ζ = 1): The system returns to equilibrium as quickly as possible without oscillating. This is often desired for the damping of systems such as doors.
// Underdamped (0 < ζ < 1): The system oscillates (at reduced frequency compared to the undamped case) with the amplitude gradually decreasing to zero.
// Undamped (ζ = 0): The system oscillates at its natural resonant frequency (ωo).
//l = kd/(2*sqrt(m*ks));
// damping coefficient Kg/s
// float kd = 10*2*sqrt(mass*ks);
#define kd 2*sqrt(mass*ks)
// float kd = 0.5* 2*sqrt(mass*ks);
// float kd = 0.0* 2*sqrt(mass*ks);
// float kd=0.0001;
// preserving volume coefficient
#define  kv 0.05

//s
#define initialPace 0.001
//s
#define stimulusPeriod 4.0

#define volIni dx*dx*dx
typedef struct str_elem{
    float pos[DIM];
    float vel[DIM];
    int stateV;
    int stateF;
    float cellTime;
    int paceMaker;
    int condition;
}typ_elem;
int contFreq=0;
float areaIni=0.0;

/**
 */
void iniElement(typ_elem *elem_new, typ_elem *elem_old, int i, int j, int k){
	for(int l=0;l<DIM;l++){
		elem_new->vel[l] = 0.0;
		elem_old->vel[l] = 0.0;
	}
	elem_new->pos[0] = i*dx;
	elem_new->pos[1] = j*dx;
	elem_new->pos[2] = k*dx;
	
	elem_old->pos[0] = i*dx;
	elem_old->pos[1] = j*dx;
	elem_old->pos[2] = k*dx;
	
	elem_old->stateV	= 0;
	elem_old->stateF	= 0;
	elem_old->cellTime	= 0.0;
	elem_old->paceMaker	= 0;
	elem_old->condition	= HEALTHY;
	
	elem_new->stateV	= 0;
	elem_new->stateF	= 0;
	elem_new->cellTime	= 0.0;
	elem_new->paceMaker	= 0;
	elem_new->condition	= HEALTHY;
}
/**
 * 
 */
__device__ float norm (float v[DIM]){
	return sqrt(pow(v[0],2) + pow(v[1],2) + pow(v[2],2));
}
/**
 */
__device__ float dotProduct(float a[DIM], float b[DIM]){
	return a[0]*b[0] + a[1]*b[1] + a[2]*b[2];
}
__device__ float det3(float a[DIM][DIM]){
	return a[0][0]*((a[1][1]*a[2][2]) - (a[2][1]*a[1][2])) -a[0][1]*(a[1][0]*a[2][2] - a[2][0]*a[1][2]) + a[0][2]*(a[1][0]*a[2][1] - a[2][0]*a[1][1]);
}

/**
 */
__device__ float getVolCube(float a[DIM], float b[DIM], float c[DIM], float d[DIM]){
	float m[DIM][DIM];
	for( int l=0; l<DIM; l++){
		m[0][l] = b[l] - a[l];
		m[1][l] = c[l] - a[l];
		m[2][l] = d[l] - a[l];
	}
	return fabs(det3(m));
}
/**
 */
__device__ float getDistance(float a[DIM], float b[DIM]){
	float aux[DIM];
	aux[0] = b[0] - a[0];
	aux[1] = b[1] - a[1];
	aux[2] = b[2] - a[2];
	return norm(aux);
}
/**
 */
__device__ void getDampingForce(int i,int j, int k, typ_elem* mesh, float force[DIM]){
	
	for(int l=0; l<DIM; l++)
		force[l] += -(mesh[get1DIndex(i,j,k)].vel[l]*kd);
}
/**
 */
__device__ float getVolRestriction(int i, int i1, int j, int j1, int k, int k1, typ_elem* mesh){
	float vol = getVolCube(		mesh[get1DIndex(i,j,k)].pos, 
					mesh[get1DIndex(i1,j,k)].pos, 
					mesh[get1DIndex(i,j1,k)].pos,
					mesh[get1DIndex(i,j,k1)].pos);
	return (vol - volIni)/volIni;
}
/**
 */
__device__ void getBaricenter(int i, int i1, int j, int j1, int k, int k1, typ_elem* mesh, float b[DIM]){
	for(int l=0;l<DIM;l++){
		b[l] = (
			mesh[get1DIndex(i,j,k)].pos[l]		+ mesh[get1DIndex(i1,j,k)].pos[l]	+ mesh[get1DIndex(i1,j1,k)].pos[l]	+ mesh[get1DIndex(i,j1,k)].pos[l] +
			mesh[get1DIndex(i,j1,k1)].pos[l]	+ mesh[get1DIndex(i,j,k1)].pos[l]	+ mesh[get1DIndex(i1,j,k1)].pos[l]	+ mesh[get1DIndex(i1,j1,k1)].pos[l]
			)/8.0;
	}
}
/**
 */
__device__ float getNeighborCube_VolPreservingForce(int i, int i1, int j, int j1, int k, int k1, typ_elem* mesh, float force[DIM]){
	float r = getVolRestriction(i, i1, j, j1, k, k1, mesh);
	float b[DIM];	float dir;
	getBaricenter(i, i1, j, j1, k, k1, mesh, b);
	int i1d = get1DIndex(i,j,k);
	float n =getDistance( b, mesh[i1d].pos);
	for(int l=0;l<DIM;l++){
		dir		= (mesh[i1d].pos[l] -b[l])/n;
		force[l]	+= -kv*r*dir;
	}
}

/**
 */
__device__ void getHookeForce(int i, int j, int k, typ_elem* mesh, float force[DIM], int ii, int jj, int kk){
	float delta[DIM];
	int i1d		= get1DIndex(i,j,k);
	int i1d2	= get1DIndex(ii,jj,kk);
	delta[0] = (ii-i)*dx - (mesh[i1d2].pos[0] - mesh[i1d].pos[0]); 
	delta[1] = (jj-j)*dx - (mesh[i1d2].pos[1] - mesh[i1d].pos[1]); 
	delta[2] = (kk-k)*dx - (mesh[i1d2].pos[2] - mesh[i1d].pos[2]);
	for(int l=0; l<DIM; l++){
		force[l] += -delta[l]*ks;
	}
}

/**
 */
__device__ float getMeshVol(typ_elem* mesh){
	float vol = 0.0, hm=0.0;
	int cont=0;
	float a1=0.0, a2=0.0;
	for(int i=0; i<X_SIZE-1; i++){
		for(int j=0; j<Y_SIZE-1; j++){
			//Obtem a area da fave z=0
			a1 += getDistance(mesh[get1DIndex(i,j,0)].pos, mesh[get1DIndex(i+1,j,0)].pos)*getDistance(mesh[get1DIndex(i,j,0)].pos, mesh[get1DIndex(i,j+1,0)].pos);
			//Obtem a area da fave z=Z_SIZE-1
			a2 += getDistance(mesh[get1DIndex(i,j,Z_SIZE-1)].pos, mesh[get1DIndex(i+1,j,Z_SIZE-1)].pos)*getDistance(mesh[get1DIndex(i,j,Z_SIZE-1)].pos, mesh[get1DIndex(i,j+1,Z_SIZE-1)].pos);
			// 	encontra a altura media
			hm += getDistance(mesh[get1DIndex(i,j,0)].pos, mesh[get1DIndex(i,j,Z_SIZE-1)].pos);
			cont++;
		}
	}

	hm = hm/cont;
	vol = ((a1+a2)/2) *hm;
	return vol;
	
}
/**
 * return the force in Newton 
*/
__device__ float getF(typ_elem* elem){
	float f=0.0;
	if(elem->condition==HEALTHY){
		switch(elem->stateF){
			case 0:
				f=0.0;
			break;
			case 1:
				f = (elem->cellTime-0.050f)/0.080f;
			break;
			case 2:
				f = 1.0;
			break;
			case 3:
				f= 1.0- (elem->cellTime-0.230f)/0.080f;
			break;
			default:
				f= 0.0;
		}
	}
	if(elem->condition==ISCHEMIC){
		switch(elem->stateF){
			case 0:
				f=0.0;
			break;
			case 1:
				f = (elem->cellTime-0.050f)/0.040f;
			break;
			case 2:
				f = 1.0;
			break;
			case 3:
				f= 1.0- (elem->cellTime-0.140f)/0.035f;
			break;
			default:
				f= 0.0;
		}
	}
	if(elem->condition==DEAD) f=0.0;
	return f*tForce;
}
/**
 * return V in miliVolts
 */
__device__ float getV(typ_elem* elem){
	if(elem->condition==HEALTHY){
		switch(elem->stateV){
			case 0:
				return -90.0;
			break;
			case 1:
				return  20.0 + ((elem->cellTime-0.000)/0.050)*(-20.0f);
			break;
			case 2:
				return   0.0 + ((elem->cellTime-0.050)/0.080)*(-25.0f);
			break;
			case 3:
				return -25.0 + ((elem->cellTime-0.130)/0.080)*(-25.0f);
			break;
			case 4:
				return -50.0 + ((elem->cellTime-0.210)/0.050)*(-40.0f);
			break;
			default:
				return -90.0;
		}
	}
	if(elem->condition==ISCHEMIC){
		switch(elem->stateV){
			case 0:
				return -70.0;
			break;
			case 1:
				return  0.0 + ((elem->cellTime-0.000)/0.050)*(-40.0f);
			break;
			case 2:
				return -40.0 + ((elem->cellTime-0.050)/0.040)*(-20.0f);
			break;
			case 3:
				return -60.0 + ((elem->cellTime-0.090)/0.025)*(-5.0f);
			break;
			case 4:
				return -65.0 + ((elem->cellTime-0.115)/0.010)*(-5.0f);

			break;
			default:
				return -70.0;
		}
	}
	if(elem->condition==DEAD){
		return 0.0;
	}
}
/**
 * */
__device__ void incStates(typ_elem* elem, float dt, typ_elem* elem_old){
	if(elem_old->condition!=DEAD){
		elem->cellTime = elem_old->cellTime + dt;
	}
	//se o elemento é saudável
	if(elem_old->condition==HEALTHY){
		switch(elem_old->stateV){
			case 1:
				if(elem_old->cellTime >= 0.050)
					elem->stateV = elem_old->stateV+1;
			break;
			case 2:
				if(elem_old->cellTime >= 0.130)
					elem->stateV = elem_old->stateV+1;
			break;
			case 3:
				if(elem_old->cellTime >= 0.210)
					elem->stateV = elem_old->stateV+1;
			break;
			case 4:
				if(elem_old->cellTime >= 0.260){
					elem->stateV = 0;
				}
			break;
			
		}
		
		switch(elem_old->stateF){
			case 0:
				if(elem_old->cellTime >= 0.050f)
					elem->stateF = elem_old->stateF+1;
			break;
			case 1:
				if(elem_old->cellTime >= 0.130f)
					elem->stateF = elem_old->stateF+1;
			break;
			case 2:
				if(elem_old->cellTime >= 0.230)
					elem->stateF = elem_old->stateF+1;
			break;
			case 3:
				if(elem_old->cellTime >= 0.310){
					elem->stateF = 0;
					elem->cellTime = 0.0f;
				}
			break;
		}
	}
	//se o elemento é isquemico
	if(elem_old->condition==ISCHEMIC){
		switch(elem_old->stateV){
			case 1:
				if(elem_old->cellTime >= 0.050)
					elem->stateV = elem_old->stateV+1;
			break;
			case 2:
				if(elem_old->cellTime >= 0.090)
					elem->stateV = elem_old->stateV+1;
			break;
			case 3:
				if(elem_old->cellTime >= 0.115)
					elem->stateV = elem_old->stateV+1;
			break;
			case 4:
				if(elem_old->cellTime >= 0.125){
					elem->stateV = 0;
				}
			break;
			
		}
		
		switch(elem_old->stateF){
			case 0:
				if(elem_old->cellTime >= 0.050)
					elem->stateF = elem_old->stateF+1;
			break;
			case 1:
				if(elem_old->cellTime >= 0.090)
					elem->stateF = elem_old->stateF+1;
			break;
			case 2:
				if(elem_old->cellTime >= 0.140)
					elem->stateF = elem_old->stateF+1;
			break;
			case 3:
				if(elem_old->cellTime >= 0.175){
					elem->stateF = 0;
					elem->cellTime = 0.0f;
				}
			break;
		}
	}
	//se o elemento está morto, não faz nada!
	

}
__device__ int isStimulationTime(float t, float dt){
	if(((t-initialPace-((int)((t-initialPace)/stimulusPeriod))*stimulusPeriod) <= dt)){
// 		printf("estimulo %f\n", t);
		return 1;
	}else{
		return 0;
	}
}
/**
 */
__device__ void cellActivation(typ_elem* elem){
	elem->stateV	= 1;
        elem->stateF	= 0;
        elem->cellTime	= 0.0f;
}

float max(float v[DIM]){
	float maior=v[0];
	for (int l=1; l<DIM; l++){
		if(v[l]>maior)
			maior = v[l];
	}
	return maior;
}
/**
 */
__device__ float getPropagationTime(typ_elem* elem, typ_elem* neighbor, float *mdir){
	//get the vector between the elements position
	float v[DIM];
	for(int l=0;l<DIM;l++){
		v[l] = neighbor->pos[l] - elem->pos[l];
	}
	//obtem a distancia entre o elemento e o vizinho
	float s = norm(v);
	float vn[DIM];
	vn[0] = v[0]/s;
	vn[1] = v[1]/s;
	vn[2] = v[2]/s;
	
	float v_fiber	= 0.70;
	float v_sheet	= 0.45;
	float v_nsheet	= 0.45;
	//encontra a velocidade na fibra
	float aux[DIM][DIM];
	//fiber
	aux[0][0]	= mdir[0]*v_fiber;
	aux[1][0]	= mdir[1]*v_fiber;
	aux[2][0]	= mdir[2]*v_fiber;
	//sheet
	aux[0][1]	= mdir[3]*v_sheet;
	aux[1][1]	= mdir[4]*v_sheet;
	aux[2][1]	= mdir[5]*v_sheet;
	//normal sheet
	aux[0][2]	= mdir[6]*v_nsheet;
	aux[1][2]	= mdir[7]*v_nsheet;
	aux[2][2]	= mdir[8]*v_nsheet;
	
	//encontra a velocidade entre os elementos
	float prEsc[DIM];
	//produto escalar de v e a velocidade na fibra
	prEsc[0] = aux[0][0]*vn[0] + aux[1][0]*vn[1] + aux[2][0]*vn[2];
	//produto escalar de v e a velocidade na sheet
	prEsc[1] = aux[0][1]*vn[0] + aux[1][1]*vn[1] + aux[2][1]*vn[2];
	//produto escalar de v e a velocidade na normal sheet
	prEsc[2] = aux[0][2]*vn[0] + aux[1][2]*vn[1] + aux[2][2]*vn[2];
	float vel_ = norm(prEsc);	
	return s/vel_;
	
}

/**
 */
__device__ void getAPForce(typ_elem* elem, typ_elem* neighbor, float *mdir, float force[DIM]){
	//get the vector between the elements position
	float v[DIM];
	for(int l=0;l<DIM;l++){
		v[l] = neighbor->pos[l] - elem->pos[l];
	}
	
	float s = norm(v);
	//normaliza o vetor
	v[0] = v[0]/s;
	v[1] = v[1]/s;
	v[2] = v[2]/s;
	//encontra o produto escalar entre v e as direções
	float aux[DIM];
	aux[0] = v[0]*mdir[0] + v[1]*mdir[1] + v[2]*mdir[2];
	aux[1] = v[0]*mdir[3] + v[1]*mdir[4] + v[2]*mdir[5];
	aux[2] = v[0]*mdir[6] + v[1]*mdir[7] + v[2]*mdir[8];
		
	float f = getF(elem);
	force[0] += aux[0]*f*mdir[0];
	force[1] += aux[1]*f*mdir[1];
	force[2] += aux[2]*f*mdir[2];
}

/**
 */
__device__ void cellularAutomaton(typ_elem* elem_old, typ_elem* elem_new, float t, float dt, int countStimulingNeighbors){
	//0 pode ser estimulado
	//1 pode estimular os vizinhos
	//2 pode estimular os vizinhos
	//3 não pode ser estimulado ou estimular
	//4 pode ser estimulado por 2 vizinhos ou mais mas não pode estimular
	//se o elemento está nos estados 0 e 4, onde o mesmo pode ser estimulado
	
	if( elem_old->stateV==0 || elem_old->stateV==4){
		if(elem_old->stateV==0 && countStimulingNeighbors>=1){
			cellActivation(elem_new);
		}else if(elem_old->stateV==4 && countStimulingNeighbors>=2){
			cellActivation(elem_new); 
		//se eh pacemaker e esta no momento de estimular
		}else if(elem_old->paceMaker==1 && isStimulationTime(t, dt)==1){
			cellActivation(elem_new);
		}else
// 		if(elem_old->stateF!= 0){
			incStates(elem_new, dt, elem_old);
// 		}
			

	}else{
		incStates(elem_new, dt, elem_old);
		if(elem_old->paceMaker==1 && isStimulationTime(t, dt)==1){
			cellActivation(elem_new);
		}
		
		
	}
}

/**
 * simulate one iteration
 */
__global__ void simulate(typ_elem *mesh_old, typ_elem *mesh_new, float t, float dt, float *mDirection){
	
	uint i = (blockIdx.x * blockDim.x) + threadIdx.x;
	uint j = (blockIdx.y * blockDim.y) + threadIdx.y;
	uint k = (blockIdx.z * blockDim.z) + threadIdx.z;
	float force[DIM];
	force[0]=0.0;
	force[1]=0.0;
	force[2]=0.0;
	int i1D = get1DIndex(i,j,k);
	if((i>=0 && i<X_SIZE) && (j>=0 && j<Y_SIZE) && (k>=0 && k<Z_SIZE)){
		//verifica quantos vizinhos estão nos estados 1 e 2, que podem estimular vizinhos
		//também considera o tempo de ativação da célula e o tempo de viagem do estímulo entre os elementos
		int countStimulingNeighbors=0;
		
		//perform a search to find all neighbors
		for(int ii=i-1;ii<=i+1;ii++){
			if(ii!=-1 && ii!=X_SIZE){
				for(int jj=j-1;jj<=j+1;jj++){
					if(jj!=-1 && jj!=Y_SIZE){
						for(int kk=k-1;kk<=k+1;kk++){
							if(kk!=-1  && kk!=Z_SIZE){
								//this is for not computing stuff when the neighbor=element
								if(!(kk==k && jj==j && ii==i)){
									//gets the action potential force
									
									getAPForce(&(mesh_old[i1D]), &(mesh_old[get1DIndex(ii,jj,kk)]), mDirection, force);
								
									//verifica o estado
									if(	( mesh_old[get1DIndex(ii,jj,kk)].stateV==1 || mesh_old[get1DIndex(ii,jj,kk)].stateV==2) &&
											(mesh_old[i1D].stateV==0 || mesh_old[i1D].stateV==4)
									){
										//verifica se estimulo é capaz de percorrer a distanica entre os elementos
										if(mesh_old[get1DIndex(ii,jj,kk)].cellTime >= getPropagationTime(&(mesh_old[i1D]), &(mesh_old[get1DIndex(ii,jj,kk)]), mDirection)){
											countStimulingNeighbors++;
											
										}
									}
								
									//gets force by Hooke's law
									getHookeForce(i, j, k, mesh_old, force, ii, jj, kk);
									//volume preserving force
									if(kv!=0.0){
										if(kk!=k && jj!=j && ii!=i)
											getNeighborCube_VolPreservingForce(i, ii, j, jj, k, kk, mesh_old, force);
									}
								}
							}
						}
					}
				}
			}
		}
		//get the damping force on this element
		getDampingForce(i, j, k, mesh_old, force);
		//cellular automaton : this changes the elements states
		cellularAutomaton(&(mesh_old[i1D]), &(mesh_new[i1D]), t, dt, countStimulingNeighbors);
		for(int l=0;l<DIM;l++){
			mesh_new[i1D].pos[l] = mesh_old[i1D].pos[l] + (mesh_old[i1D].vel[l])*dt;
			mesh_new[i1D].vel[l] = mesh_old[i1D].vel[l] + (force[l]/mass)*dt;
		}
	
	}
}

__global__ void stepAhead(typ_elem* mesh_new, typ_elem* mesh_old, float dt, float *dev_vol){
	int i1d;
	for(int i=0;i<X_SIZE;i++){
		for(int j=0;j<Y_SIZE;j++){
			for(int k=0;k<Z_SIZE;k++){
				i1d = get1DIndex(i,j,k);
				mesh_old[i1d].stateV	= mesh_new[i1d].stateV;
				mesh_old[i1d].stateF	= mesh_new[i1d].stateF;
				mesh_old[i1d].cellTime	= mesh_new[i1d].cellTime;
			}
		}
	}
	typ_elem*aux	 = mesh_new;
	mesh_new = mesh_old;
	mesh_old = aux;
	*dev_vol = getMeshVol(mesh_old);
}
/**
 */
void timeIntegration()
{	
	float *mDirection;
	int size = sizeof(typ_elem)*X_SIZE*Y_SIZE*Z_SIZE ;
	int size2 = sizeof(float)*DIM*DIM;
	
	mDirection = (float*)malloc(size2);
	//fiber
	mDirection[0] = 1.0;
	mDirection[1] = 0.0;
	mDirection[2] = 0.0;
	//sheet
	mDirection[3] = 0.0;
	mDirection[4] = 1.0;
	mDirection[5] = 0.0;
	//
	mDirection[6] = 0.0;
	mDirection[7] = 0.0;
	mDirection[8] = 1.0;
	
	typ_elem *mesh_new = (typ_elem*)malloc(size);
	typ_elem *mesh_old = (typ_elem*)malloc(size);
	int i1d;
	for(int i=0;i<X_SIZE;i++){
		for(int j=0;j<Y_SIZE;j++){
			for(int k=0;k<Z_SIZE;k++){
				i1d = get1DIndex(i, j, k);
				iniElement(&(mesh_new[i1d]), &(mesh_old[i1d]), i, j, k);
				
			}
		}
	}
	
	mesh_new[0].paceMaker=1;
	mesh_old[0].paceMaker=1;
	typ_elem *device_mesh_new;
	typ_elem *device_mesh_old;
	float* fiber_device;
	
	hipMalloc( (void**)&device_mesh_new,	size );
	hipMalloc( (void**)&device_mesh_old,	size );
	hipMalloc( (void**)&fiber_device,	size2 );
	// copy host memory to device
	hipMemcpy(device_mesh_new,	mesh_new,	size,	hipMemcpyHostToDevice);
	hipMemcpy(device_mesh_old,	mesh_old,	size,	hipMemcpyHostToDevice);
	hipMemcpy(fiber_device,	mDirection,	size2,	hipMemcpyHostToDevice);
	
// 	cudaThreadSetLimit(cudaLimitMallocHeapSize, 100*1024*1024*1024);
	
	dim3 threadsPerBlock(3, 3, 3);

	dim3 numBlocks(	X_SIZE/threadsPerBlock.x,
			Y_SIZE/threadsPerBlock.y, 
			Z_SIZE/threadsPerBlock.z);

	float *dev_v;
	float host_v;
	hipMalloc( (void**)&dev_v, sizeof(float) ) ;
	
	int i=1, j=1, k=1, cont=0;
	
// 	float areaMeshIni  = getMeshArea(pos_old);
	//simulation time in s
	float tfinal= 1.0;
	//time step in s : 0,1ms
	float dt=1.0e-4;
	float t=0;
	
	
	while(t<=tfinal){
		simulate<<<numBlocks,threadsPerBlock>>>( device_mesh_old, device_mesh_new, t, dt, fiber_device);
		stepAhead<<<1,1>>>( device_mesh_old, device_mesh_new, dt, dev_v);

		hipMemcpy( &host_v, dev_v, sizeof(float), hipMemcpyDeviceToHost);
	
		t += dt;
		cont++;
	}	
	printf("teste: %f\n", host_v);
	free(mesh_new);		free(mesh_old);
}
/**
 */
int main()
{
	timeIntegration();

}
// nvcc simulador.cu -c -o simulador.o -I/usr/local/cuda/include -L/usr/local/cuda/lib64 -w 
// g++ -o simulador simulador.o -I/usr/local/cuda/include -L/usr/local/cuda/lib64 -lcuda -lcudart
